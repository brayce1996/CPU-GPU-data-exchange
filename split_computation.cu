
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define h_Ain(i, j) h_Ain[(i)*nx +(j)]
#define h_Aout(i, j) h_Aout[(i)*nx +(j)]


// CUDA kernel. Each thread takes care of one element of c
__global__ void stencil(float *d_Ain, float *d_Aout, int nx, int ny)
{
    #define d_Ain(i, j) d_Ain[(i)*nx +(j)]
    #define d_Aout(i, j) d_Aout[(i)*nx +(j)]

    // Get our global thread ID
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    float north, south, east, west;

    if ( (i > 0 && i < (nx/2)) && (j > 0 && j < ny - 1) ) {
        float current =  d_Ain(i, j);
        north = d_Ain(i-1,j);
        south = d_Ain(i+1,j);
        east = d_Ain(i,j+1);
        west = d_Ain(i,j-1);

        d_Aout(i,j) = -4 * current + north + south + east + west;
    }
}

int main( int argc, char* argv[] )
{
    int grid_size = atoi(argv[1]);
    int enable_pinned_memory = atoi(argv[2]);

    // Size of matrix
    int nx = grid_size + 2;
    int ny = grid_size + 2;
 
    // Host input vectors
    float *h_Ain;
    
    //Host output vector
    float *h_Aout;
 
    // Device input vectors
    float *d_Ain;
 
    //Device output vector
    float *d_Aout;

    // Size, in bytes, of each vector
    size_t bytes = nx*ny*sizeof(float);

    // Allocate memory for each vector on host
    if (enable_pinned_memory) {
        hipHostMalloc((void**)&h_Ain, bytes, hipHostMallocDefault);
        hipHostMalloc((void**)&h_Aout, bytes, hipHostMallocDefault);
        // memset(h_Aout, 0, bytes);
    } else {
        h_Ain = (float*)malloc(bytes);
        h_Aout = (float*)malloc(bytes);
    }
    
    // Allocate memory for each vector on GPU
    hipMalloc(&d_Ain, bytes/2);
    hipMalloc(&d_Aout, bytes/2);

    int i,j;
    // Initialize vectors on host
    for( i = 0; i < ny; i++ ) {
        for(j = 0; j < nx; j++) {
            h_Ain(i,j) = rand();
        }
    }

    /*
    for( i = 0; i < ny; i++ ) {
        for(j = 0; j < nx; j++) {
            printf("%f \t", h_Ain(i,j));
        }
        printf("\n");
    }
    */

    float ms; // elapsed time in milliseconds
    
    // create events and streams
    hipEvent_t startEvent, stopEvent, dummyEvent;
    

    dim3 DimGrid(ceil(nx/16.0),ceil((ny/2)/16.0));
    dim3 DimBlock(16,16);

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&dummyEvent);
    // Copy host vectors to device

    hipEventRecord(startEvent,0);
  
    hipMemcpy(d_Ain, h_Ain, bytes/2, hipMemcpyHostToDevice);

    //printf("Launching kernel stencil....... \n");

    stencil<<<DimGrid,DimBlock>>>(d_Ain, d_Aout, nx, ny/2);

    //Computation on Host
    float north, south, east, west;

    if ( (i >= (nx/2) && i < nx - 1) && (j > 0 && j < ny - 1) ) {
        float current =  h_Ain(i, j);
        north = h_Ain(i-1,j);
        south = h_Ain(i+1,j);
        east = h_Ain(i,j+1);
        west = h_Ain(i,j-1);

        h_Aout(i,j) = -4 * current + north + south + east + west;
    }
    

    // Copy array back to host
    hipMemcpy(h_Aout, d_Aout, bytes/2, hipMemcpyDeviceToHost);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time for sequential transfer and execute (ms): %f\n", ms);

    
/*
    // Size, in bytes, of each vector
    size_t bytes = nx*ny*sizeof(int);
 
    // Allocate memory for each vector on host
    h_Ain = (int*)malloc(bytes);
    h_Aout = (int*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    cudaMalloc(&d_Ain, bytes);
    cudaMalloc(&d_Aout, bytes);
 
    int i,j;
    // Initialize vectors on host
    for( i = 0; i < ny; i++ ) {
        for(j = 0; j < nx; j++) {
            h_Ain[i*nx + j] = rand();
        }
    }
 
    // Copy host vectors to device
    cudaMemcpy(d_Ain, h_Ain, bytes, cudaMemcpyHostToDevice);
 
    dim3 DimGrid(ceil(nx/16.0),ceil(ny/16.0));
    dim3 DimBlock(16,16);

    printf("Ain[1] result: %d\n", h_Ain[1]);

    printf("Launching kernel stencil....... \n");
    
    // Execute the kernel
    stencil<<<DimGrid,DimBlock>>>(d_Ain, d_Aout, nx, ny);
 
    // Copy array back to host
    cudaMemcpy(h_Aout, d_Aout, bytes, cudaMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    printf("Aout[1] result: %d\n", h_Aout[1]);

*/
    //printf("%f ~~~~~~~~~ \n", h_Ain(3,4));

    // Release device memory
    hipFree(d_Ain);
    hipFree(d_Aout);

    // Release host memory
    if (enable_pinned_memory) {
        hipHostFree(h_Ain);
        hipHostFree(h_Aout);
    } else {
        free(h_Ain);
        free(h_Aout);
    }
 
    return 0;
}
